/*
 * Layer.cu
 *
 *  Created on: Mar 13, 2016
 *      Author: jamiecho
 */

#include "Layer.h"

Layer::Layer(int n):
	n(n){
	hipMalloc(&I, n*sizeof(double));
	hipMalloc(&O, n*sizeof(double));
	hipMalloc(&G, n*sizeof(double));
}

Layer::~Layer() {
	hipFree(I);
	hipFree(O);
	hipFree(G);
}

double*& Layer::transfer(double* i){
	hipMemcpy(I,i,n*sizeof(double),hipMemcpyDeviceToDevice);
	sigmoid_g(I,O,n);
	return O;
}

void Layer::setI(double* i){
	hipMemcpy(I,i,n*sizeof(double),hipMemcpyDeviceToDevice);

}

void Layer::setO(double* o){
	hipMemcpy(O,o,n*sizeof(double),hipMemcpyHostToDevice);
}


void Layer::setG(double* g){
	hipMemcpy(G,g,n*sizeof(double),hipMemcpyDeviceToDevice);
}

double*& Layer::getI(){
	return I;
}

double*& Layer::getO(){
	return O;
}

double*& Layer::getG(){
	return G;
}

int Layer::size(){
	return n;
}
